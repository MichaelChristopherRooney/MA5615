
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <string>

void custom_error_check(hipError_t result, std::string err_str){
	if(result != hipSuccess){
		std::cout << err_str << "\n";
		std::cout << "Error code: " << result << "\n";
		exit(1);
	}	
}

// Taken from provided sample code
int find_best_device() {
	int i,n,best,bestNumberOfMultiprocessors;
	hipGetDeviceCount(&n);
	int numberOfCUDAcoresForThisCC=0;
	struct hipDeviceProp_t x;
	best=-1;
	bestNumberOfMultiprocessors=-1;
	for (i=0;i<n;i++) {
		hipGetDeviceProperties(&x, i);
		switch (x.major) {
			case 1:	// Tesla / T10
				numberOfCUDAcoresForThisCC=8;
				break;
			case 2:	// Fermi
				numberOfCUDAcoresForThisCC=32;
				break;
			case 3:	// Kepler
				numberOfCUDAcoresForThisCC=192;
				break;
			case 5:	// Maxwell
				numberOfCUDAcoresForThisCC=128;
				break;
			case 6:	// Pascal
				switch (x.minor) {
					case 0: // GP100, 64 cuda cores per SM - 7.0 should be prefered over 7.1
						numberOfCUDAcoresForThisCC=64;
						break;
					case 1: // GP102, GP104, GP106, GP107, 128 cuda cores per SM
						numberOfCUDAcoresForThisCC=128;
						break;
					default: // Unknown
						numberOfCUDAcoresForThisCC=0;
						break;
				}
				numberOfCUDAcoresForThisCC=128;
				break;
			case 7:	// Volta
				numberOfCUDAcoresForThisCC=64;
				break;
			default: // Unknown
				numberOfCUDAcoresForThisCC=0;
				break;
		}
		if (x.multiProcessorCount>bestNumberOfMultiprocessors*numberOfCUDAcoresForThisCC) {
			best=i;
			bestNumberOfMultiprocessors=x.multiProcessorCount*numberOfCUDAcoresForThisCC;
		}
	}
	return best;
}

__device__ float device_exp_integral_float(int n, const float x){
	const int max_iter = 2000000000;
	const float e_const_float = 0.5772156649015329f;
	const float epsilon_float = 1.E-30f;
	int i, ii, nm1 = n-1;
	float a, b, c, d, del, fact, h, psi, ans=0.0;
	if(n == 0){
		return expf(-x)/x;
	}
	if(x > 1.0f){
		b = x + n;
		c = 3.402823E38;
		d = 1.0f/b;
		h=d;
		for(i = 1; i <= max_iter; i++){
			a=-i*(nm1+i);
			b+=2.0;
			d=1.0/(a*d+b);
			c=b+a/c;
			del=c*d;
			h*=del;
			if (fabsf(del-1.0)<=epsilon_float) {
				return h*expf(-x);
			}
		}
                return h*expf(-x);
	} else {
		ans=(nm1!=0 ? 1.0/nm1 : -log(x)-e_const_float); // First term
		fact=1.0;
		for (i=1;i<=max_iter;i++) {
			fact*=-x/i;
			if (i != nm1) {
				del = -fact/(i-nm1);
			} else {
				psi = -e_const_float;
				for (ii=1;ii<=nm1;ii++) {
					psi += 1.0/ii;
				}
				del=fact*(-log(x)+psi);
			}
			ans+=del;
			if (fabsf(del)<fabsf(ans)*epsilon_float) return ans;
		}
		return ans;
	}
}

// Note: need to use j-1 as an index since j starts at 1 rather than 0
// Same with using idx rather than my_n, as the minimum my_n is 1 rather than 0.
// TODO: move variables (like division) to constant memory to save registers
__global__ void device_part_float(
		const float division, const int n, const int num_samples, const float a, 
		float *device_float_results
	){
	const int idx=blockIdx.x*blockDim.x+threadIdx.x;
	const int my_n = idx + 1;
	if(my_n > n){
		return;
	}
	const int offset = (idx * num_samples);
	float x;
	float4 f_res;
	int j;
	for(j = 1; j <= num_samples - 4; j = j + 4){
		x = a+(j*division);
		f_res.x = device_exp_integral_float(my_n, x);
		x = a+((j+1)*division);
		f_res.y = device_exp_integral_float(my_n, x);
		x = a+((j+2)*division);
		f_res.z = device_exp_integral_float(my_n, x);
		x = a+((j+3)*division);
		f_res.w = device_exp_integral_float(my_n, x);
		*((float4 *)&(device_float_results[offset + (j-1)])) = f_res;
	}
	// Handle any remaining work if num_samples does not divide evenly by 4
	for(; j <= num_samples; j++){
                x = a+(j*division);
                f_res.x = device_exp_integral_float(my_n, x);
		device_float_results[offset + (j-1)] = f_res.x;
	}
}

__device__ double device_exp_integral_double(int n, const double x){
	const int max_iter = 2000000000;
	const double e_const_double = 0.5772156649015329;
	const double epsilon_double = 1.E-30;
	int i, ii, nm1 = n-1;
	double a, b, c, d, del, fact, h, psi, ans=0.0;
	if(n == 0){
		return exp(-x)/x;
	}
	if(x > 1.0f){
		b = x + n;
		c = 3.402823E38;
		d = 1.0f/b;
		h=d;
		for(i = 1; i <= max_iter; i++){
			a=-i*(nm1+i);
			b+=2.0;
			d=1.0/(a*d+b);
			c=b+a/c;
			del=c*d;
			h*=del;
			if (fabs(del-1.0)<=epsilon_double) {
				return h*exp(-x);
			}
		}
                return h*exp(-x);
	} else {
		ans=(nm1!=0 ? 1.0/nm1 : -log(x)-e_const_double); // First term
		fact=1.0;
		for (i=1;i<=max_iter;i++) {
			fact*=-x/i;
			if (i != nm1) {
				del = -fact/(i-nm1);
			} else {
				psi = -e_const_double;
				for (ii=1;ii<=nm1;ii++) {
					psi += 1.0/ii;
				}
				del=fact*(-log(x)+psi);
			}
			ans+=del;
			if (fabs(del)<fabs(ans)*epsilon_double) return ans;
		}
		return ans;
	}
}

// Note: need to use j-1 as an index since j starts at 1 rather than 0
// Same with using idx rather than my_n, as the minimum my_n is 1 rather than 0.
// TODO: move variables (like division) to constant memory to save registers
__global__ void device_part_double(
		const double division, const int n, const int num_samples, const double a, 
		double *device_double_results
	){
	const int idx=blockIdx.x*blockDim.x+threadIdx.x;
	const int my_n = idx + 1;
	if(my_n > n){
		return;
	}
	const int offset = (idx * num_samples);
	double x;
	double4 d_res;
	int j;
	for(j = 1; j <= num_samples - 4; j = j + 4){
		x = a+(j*division);
		d_res.x = device_exp_integral_double(my_n, x);
		x = a+((j+1)*division);
		d_res.y = device_exp_integral_double(my_n, x);
		x = a+((j+2)*division);
		d_res.z = device_exp_integral_double(my_n, x);
		x = a+((j+3)*division);
		d_res.w = device_exp_integral_double(my_n, x);
		*((double4 *)&(device_double_results[offset + (j-1)])) = d_res;
	}
	// Handle any remaining work if num_samples does not divide evenly by 4
	for(; j <= num_samples; j++){
                x = a+(j*division);
                d_res.x = device_exp_integral_double(my_n, x);
		device_double_results[offset + (j-1)] = d_res.x;
	}
}

// TODO: fix up events and record to a struct rather than printing info here
// Assuming this is run on CUDA01 it does the following:
//	1) The float code is run on the GTX 780
//	2) The double code is run on the Tesla K40c
extern void do_cuda_part(
		double a, double b, unsigned int n, unsigned int num_samples, 
		int block_size, float **float_results, double **double_results
	){
	// Data needed by both kernels
	unsigned int size = n * num_samples;
	double division = (b-a)/(double)num_samples;
	dim3 dimBlock(block_size);
	dim3 dimGrid ( (n/dimBlock.x) + (!(n%dimBlock.x)?0:1) );
	float t = 0;

	// Set up device for double version and device for float version
	int double_device_id = find_best_device();
	int float_device_id = double_device_id == 0 ? 1 : 0;

	// Set up events
	hipSetDevice(float_device_id);
	hipEvent_t f_start, f_stop;
	hipEventCreate(&f_start);
	hipEventCreate(&f_stop);
	hipSetDevice(double_device_id);
	hipEvent_t d_start, d_stop;
	hipEventCreate(&d_start);
	hipEventCreate(&d_stop);

	// Set up streams
	hipSetDevice(float_device_id);
	hipStream_t float_stream;
	hipStreamCreate(&float_stream);
	hipSetDevice(double_device_id);
	hipStream_t double_stream;
	hipStreamCreate(&double_stream);

	// Allocate space on float device for results
	float *device_float_results;
	hipSetDevice(float_device_id);
	hipEventRecord(f_start, float_stream);
	custom_error_check(
		hipMalloc((void **) &device_float_results, size * sizeof(float)), 
		"Failed to allocate float result buffer on device."
	);
	hipEventRecord(f_stop, float_stream);
	hipEventSynchronize(f_stop);
	hipEventElapsedTime(&t, f_start, f_stop);
	printf("Float alloc: %f\n", t);

	// Allocate space on double device for results
	double *device_double_results;
	hipSetDevice(double_device_id);
	hipEventRecord(d_start, double_stream);
	custom_error_check(
		hipMalloc((void **) &device_double_results, size * sizeof(double)), 
		"Failed to allocate double result buffer on device."
	);
	hipEventRecord(d_stop, double_stream);
	hipEventSynchronize(d_stop);
	hipEventElapsedTime(&t, d_start, d_stop);
	printf("Double alloc: %f\n", t);

	// Now run the kernels on different streams
	hipSetDevice(float_device_id);
	hipEventRecord(f_start, float_stream);
	device_part_float<<<dimGrid,dimBlock, 0, float_stream>>>((float)division, n, num_samples, (float)a, device_float_results);
	hipEventRecord(f_stop, float_stream);
	hipSetDevice(double_device_id);
	hipEventRecord(d_start, double_stream);
	device_part_double<<<dimGrid,dimBlock, 0, double_stream>>>(division, n, num_samples, a, device_double_results);
	hipEventRecord(d_stop, double_stream);

	// Async copy results
	hipSetDevice(float_device_id);
	custom_error_check(
		hipMemcpyAsync(float_results[0], device_float_results, size * sizeof(float), hipMemcpyDeviceToHost, float_stream), 
		"Failed to copy float results from device."
	);
	hipSetDevice(double_device_id);
	custom_error_check(
		hipMemcpyAsync(double_results[0], device_double_results, size * sizeof(double), hipMemcpyDeviceToHost, double_stream), 
		"Failed to copy double results from device."
	);

	// Free device pointers
	hipSetDevice(float_device_id);
	custom_error_check(
		hipFree(device_float_results), 
		"Failed to free float results on device"
	);
	hipSetDevice(double_device_id);
	custom_error_check(
		hipFree(device_double_results), 
		"Failed to double results on device"
	);

	hipEventSynchronize(f_stop);
	hipEventElapsedTime(&t, f_start, f_stop);
	printf("Float kernel: %f\n", t);

	hipEventSynchronize(d_stop);
	hipEventElapsedTime(&t, d_start, d_stop);
	printf("Double kernel: %f\n", t);

	// Destroy streams
	hipStreamDestroy(float_stream);
	hipStreamDestroy(double_stream);
}
