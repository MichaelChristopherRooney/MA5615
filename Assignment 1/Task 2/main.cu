#include "hip/hip_runtime.h"
#include "stdlib.h"
#include "stdio.h"

#include "matrix.h"

__global__ void matrix(float *mat, int nrow, int ncol){
	int idx=blockIdx.x*blockDim.x+threadIdx.x;
	if(idx < nrow * ncol){
		printf("Thread %d has %f\n", idx, mat[idx]);
	}
}

// TODO: seed and other args
int main(int argc, char *argv[]){
	srand48(123456);
	int ncol = 5;
	int nrow = 5;
	int mat_size = sizeof(float) * nrow * ncol;
	float **mat = create_random_matrix(nrow, ncol);
	float *mat_gpu;
	hipMalloc((void **) &mat_gpu, mat_size);
	hipMemcpy(mat_gpu, mat[0], mat_size, hipMemcpyHostToDevice);
	printf("On host matrix is:\n");
	print_matrix(mat, nrow, ncol);
	int block_size = 8;
	dim3 dimBlock(block_size);
	dim3 dimGrid ( (mat_size/dimBlock.x) + (!(mat_size%dimBlock.x)?0:1) );
	matrix<<<dimGrid,dimBlock>>>(mat_gpu, nrow, ncol);
	free_matrix(mat);
	hipFree(mat_gpu);
	return 0;
}
