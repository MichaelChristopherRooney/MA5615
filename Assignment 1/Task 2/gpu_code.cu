#include "hip/hip_runtime.h"
#include "stdlib.h"
#include "stdio.h"

#include "matrix.h"

__global__ void sum_rows(float *mat, float *out, int nrow, int ncol){
	int idx=blockIdx.x*blockDim.x+threadIdx.x;
	if(idx >= nrow){
		return;
	}
	out[idx] = 0.0; // clear memory before we use it
	float result = 0.0;
	for(int i = 0; i < ncol; i++){
		int index = (ncol * idx) + i;
		//printf("Thread %d accessing %d\n", idx, index);
		result += mat[index];
	}
	out[idx] = result;
	//printf("Thread %d got %f\n", idx, result);
}

__global__ void sum_columns(float *mat, float *out, int nrow, int ncol){
	int idx=blockIdx.x*blockDim.x+threadIdx.x;
	if(idx >= ncol){
		return;
	}
	out[idx] = 0.0; // clear memory before we use it
	float result = 0.0;
	for(int i = 0; i < nrow; i++){
		int index = (ncol * i) + idx;
		//printf("Thread %d accessing %d\n", idx, index);
		result += mat[index];
	}
	out[idx] = result;
	//printf("Thread %d got %f\n", idx, result);
}

extern "C" void do_gpu_sums(float **mat, float *row_sum_vec, float *col_sum_vec, int nrow, int ncol){
	int mat_size = sizeof(float) * nrow * ncol;
	float *mat_gpu;
	float *col_sum_vec_gpu;
	float *row_sum_vec_gpu;
	hipMalloc((void **) &mat_gpu, mat_size);
	hipMalloc((void **) &row_sum_vec_gpu, nrow * sizeof(float));
	hipMalloc((void **) &col_sum_vec_gpu, ncol * sizeof(float));
	hipMemcpy(mat_gpu, mat[0], mat_size, hipMemcpyHostToDevice);
	int block_size = 8;
	dim3 dimBlock(block_size);
	dim3 dimGrid ( (mat_size/dimBlock.x) + (!(mat_size%dimBlock.x)?0:1) );
	sum_rows<<<dimGrid,dimBlock>>>(mat_gpu, row_sum_vec_gpu, nrow, ncol);
	sum_columns<<<dimGrid,dimBlock>>>(mat_gpu, col_sum_vec_gpu, nrow, ncol);
	hipMemcpy(row_sum_vec, row_sum_vec_gpu, nrow * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(col_sum_vec, col_sum_vec_gpu, ncol * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(mat_gpu);
	hipFree(row_sum_vec);
	hipFree(col_sum_vec);
}

