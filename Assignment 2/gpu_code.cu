#include "hip/hip_runtime.h"
#include "stdlib.h"
#include "stdio.h"
#include <string>
#include <iostream>

#include "grid.h"

void custom_error_check(hipError_t result, std::string err_str){
	if(result != hipSuccess){
		std::cout << err_str << "\n";
		std::cout << "Error code: " << result << "\n";
		exit(1);
	}	
}

// Each thread takes one row
// Uses mostly global memory
__global__ void cuda_do_grid_iterations_naive_ver(DATA_TYPE *grid_gpu_1, DATA_TYPE *grid_gpu_2, int nrow, int ncol, int num_iter){
	int idx=blockIdx.x*blockDim.x+threadIdx.x;
        long long row_offset = idx * ncol;
        DATA_TYPE *cur = grid_gpu_1;
        DATA_TYPE *next = grid_gpu_2;
        if(idx >= nrow){
                return;
        }
        for(int i = 0; i < num_iter; i++){
                for(long long n = 2; n < ncol; n++){
                        next[row_offset+n] = 0.15*(cur[row_offset+n-2]);
                        next[row_offset+n] += 0.65*(cur[row_offset+n-1]);
                        next[row_offset+n] += (cur[row_offset+n]);
                        if(n == ncol - 2){
                                next[row_offset+n] += 1.35*(cur[row_offset+n+1]);
                                next[row_offset+n] += 1.85*(cur[row_offset]);
                        } else if(n == ncol - 1){
                                next[row_offset+n] += 1.35*(cur[row_offset]);
                                next[row_offset+n] += 1.85*(cur[row_offset+1]);
                        } else {
                                next[row_offset+n] += 1.35*(cur[row_offset+n+1]);
                                next[row_offset+n] += 1.85*(cur[row_offset+n+2]);
                        }
                        next[row_offset+n] = next[row_offset+n] / 5.0;
                }
                DATA_TYPE *temp = cur;
                cur = next;
                next = temp;
        }
	if(cur != grid_gpu_1){
		for(int n = 2; n < ncol; n++){
			grid_gpu_1[row_offset+n] = grid_gpu_2[row_offset+n];
		}
	}
}

extern "C" DATA_TYPE *do_grid_iterations_gpu_naive_ver(DATA_TYPE **grid_gpu_host, int nrow, int ncol, int block_size, int num_iter){
	long long grid_size = (long long) nrow * (long long) ncol * (long long) sizeof(DATA_TYPE);
	DATA_TYPE *grid_gpu_device_1;
	DATA_TYPE *grid_gpu_device_2;
	custom_error_check(hipMalloc((void **) &grid_gpu_device_1, grid_size), "Failed to allocate on device");
	custom_error_check(hipMalloc((void **) &grid_gpu_device_2, grid_size), "Failed to allocate on device");
	custom_error_check(hipMemcpy(grid_gpu_device_1, grid_gpu_host[0], grid_size, hipMemcpyHostToDevice), "Failed to copy data to device");
	custom_error_check(hipMemcpy(grid_gpu_device_2, grid_gpu_host[0], grid_size, hipMemcpyHostToDevice), "Failed to copy data to device");
	dim3 dimBlock(block_size);
	dim3 dimGrid ( (nrow/dimBlock.x) + (!(nrow%dimBlock.x)?0:1) );
	cuda_do_grid_iterations_naive_ver<<<dimGrid,dimBlock>>>(grid_gpu_device_1, grid_gpu_device_2, nrow, ncol, num_iter);
	custom_error_check(hipPeekAtLastError(), "Error during kernel execution");
	custom_error_check(hipMemcpy(grid_gpu_host[0], grid_gpu_device_1, grid_size, hipMemcpyDeviceToHost), "Failed to copy data FROM device");
	//custom_error_check(hipFree(grid_gpu_device_1), "Failed to free memory on device");
	custom_error_check(hipFree(grid_gpu_device_2), "Failed to free memory on device");
	return grid_gpu_device_1;
}

// See report for details on the optimisations used here.
// In a nutshell this version:
// 1) greatly reduces the number of memory operations by using registers
// 2) each row calculates its own constant values, meaning the grid does not need to copied TO the device
//	^ note the the grid must still be copied FROM the device
// 3) replace grid[i] /= 5.0 with grid[i] *= (1.0/5.0), where 1.0/5.0 is calculated once at the start.
// 4) unroll inner loop so two columns are processed per inner loop iteration
// 5) uses float2/double2 to store results that are written to memory, reducing number of memory ops
__global__ void cuda_do_grid_iterations_fast_ver(DATA_TYPE *grid_gpu, int nrow, int ncol, int num_iter){
	long long idx=blockIdx.x*blockDim.x+threadIdx.x;
	if(idx >= (nrow)){
		return;
	}
	const DATA_TYPE one_over_five = 1.0 / 5.0;
	const long long offset = idx * (long long) ncol;
	// Constants for the first row (row 0)
	const DATA_TYPE col_0_fixed_val = 0.85*(DATA_TYPE)((idx+1)*(idx+1)) / (DATA_TYPE)(nrow * nrow);
	const DATA_TYPE col_1_fixed_val= (DATA_TYPE)((idx+1)*(idx+1)) / (DATA_TYPE)(nrow * nrow);
	// Place the constants in the first two columns of the grid.
	grid_gpu[offset] = col_0_fixed_val;
	grid_gpu[offset+1] = col_1_fixed_val;
	float2 result;
	float2 next_vals;
	for(int i = 0; i < num_iter; i++){
		// Registers for the first row (row 0)
		DATA_TYPE val_n_minus_2 = col_0_fixed_val;
		DATA_TYPE val_n_minus_1 = col_1_fixed_val;
		DATA_TYPE val = grid_gpu[offset + 2];
		DATA_TYPE val_orig = val;
		DATA_TYPE val_n_plus_1 = grid_gpu[offset + 3];
		DATA_TYPE val_n_plus_2 = grid_gpu[offset + 4];
		DATA_TYPE val_next;
		for(int n = 2; n < ncol - 4; n = n + 2){
			next_vals = *(float2*)(&grid_gpu[offset+n+3]);
			// Unrolled part 0
			val += 0.15*val_n_minus_2;
			val += 0.65*val_n_minus_1;
			val_next = val_n_plus_1;
			val += 1.35*val_n_plus_1;
			val += 1.85*val_n_plus_2;
			val_n_plus_1 = val_n_plus_2;
			val_n_plus_2 = next_vals.x;
			val = val * one_over_five;
			result.x = val;
			val_n_minus_2 = val_n_minus_1;
			val_n_minus_1 = val_orig;
			val = val_next;
			val_orig = val_next;
			// Unrolled part 1
			val += 0.15*val_n_minus_2;
			val += 0.65*val_n_minus_1;
			val_next = val_n_plus_1;
			val += 1.35*val_n_plus_1;
			val += 1.85*val_n_plus_2;
			val_n_plus_1 = val_n_plus_2;
			val_n_plus_2 = next_vals.y;
			val = val * one_over_five;
			result.y = val;
			val_n_minus_2 = val_n_minus_1;
			val_n_minus_1 = val_orig;
			val = val_next;
			val_orig = val_next;
			*(float2*)(&grid_gpu[offset+n]) = result;
		}
		next_vals = *(float2*)(&grid_gpu[offset+ncol-2]);
		// Fourth last column
		val += 0.15*val_n_minus_2;
		val += 0.65*val_n_minus_1;
		val_next = val_n_plus_1;
		val += 1.35*val_n_plus_1;
		val += 1.85*val_n_plus_2;
		val_n_plus_1 = val_n_plus_2;
		val_n_plus_2 = next_vals.y;
		val = val * one_over_five;
		result.x = val;
		val_n_minus_2 = val_n_minus_1;
		val_n_minus_1 = val_orig;
		val = val_next;
		val_orig = val_next;
		// Third last column
		val += 0.15*val_n_minus_2;
		val += 0.65*val_n_minus_1;
		val_next = val_n_plus_1;
		val += 1.35*val_n_plus_1;
		val += 1.85*val_n_plus_2;
		val_n_plus_1 = val_n_plus_2;
		val = val * one_over_five;
		result.y = val;
		val_n_minus_2 = val_n_minus_1;
		val_n_minus_1 = val_orig;
		val = val_next;
		val_orig = val_next;
		*(float2*)(&grid_gpu[offset+ncol-4]) = result;
		// Second last column
		val += 0.15*val_n_minus_2;
		val += 0.65*val_n_minus_1;
		val_next = val_n_plus_1;
		val += 1.35*val_n_plus_1;
		val += 1.85*col_0_fixed_val;
		val = val * one_over_five;
		result.x = val;
		val_n_minus_2 = val_n_minus_1;
		val_n_minus_1 = val_orig;
		val = val_next;
		// Last column
		val += 0.15*val_n_minus_2;
		val += 0.65*val_n_minus_1;
		val_next = val_n_plus_1;
		val += 1.35*col_0_fixed_val;
		val += 1.85*col_1_fixed_val;
		val = val * one_over_five;
		result.y = val;
		*(float2*)(&grid_gpu[offset+ncol-2]) = result;
	}
}

extern "C" DATA_TYPE* do_grid_iterations_gpu_fast_ver(DATA_TYPE **grid_gpu_host, int nrow, int ncol, int block_size, int num_iter){
	long long grid_size = (long long) (nrow) * (long long) (ncol) * (long long) sizeof(DATA_TYPE);
	DATA_TYPE *grid_gpu_device;
	custom_error_check(hipMalloc((void **) &grid_gpu_device, grid_size), "Failed to allocate on device");
	dim3 dimBlock(block_size);
	dim3 dimGrid ( ((nrow)/dimBlock.x) + (!((nrow)%dimBlock.x)?0:1) );
	cuda_do_grid_iterations_fast_ver<<<dimGrid,dimBlock>>>(grid_gpu_device, nrow, ncol, num_iter);
	custom_error_check(hipPeekAtLastError(), "Error during kernel execution");
	custom_error_check(hipMemcpy(grid_gpu_host[0], grid_gpu_device, grid_size, hipMemcpyDeviceToHost), "Failed to copy data FROM device");
	//custom_error_check(hipFree(grid_gpu_device), "Failed to free memory on device");
	return grid_gpu_device;
}

__global__ void cuda_do_reduce_naive(DATA_TYPE *grid_device, DATA_TYPE *reduce_device, int nrow, int ncol){
	int idx=blockIdx.x*blockDim.x+threadIdx.x;
	if(idx >= nrow){
		return;
	}
	const int offset = idx * ncol;
	for(int i = 0; i < ncol; i++){
		reduce_device[idx] += grid_device[offset+i];
	}
}

extern "C" void do_reduce_naive(DATA_TYPE *grid_device, DATA_TYPE *reduce_host, int nrow, int ncol, int block_size){
	long long reduce_size = (long long) nrow * (long long) sizeof(DATA_TYPE);
	DATA_TYPE *reduce_device;
	custom_error_check(hipMalloc((void **) &reduce_device, reduce_size), "Failed to allocate on device");
	dim3 dimBlock(block_size);
	dim3 dimGrid ( ((nrow)/dimBlock.x) + (!((nrow)%dimBlock.x)?0:1) );
	cuda_do_reduce_naive<<<dimGrid,dimBlock>>>(grid_device, reduce_device, nrow, ncol);
	custom_error_check(hipPeekAtLastError(), "Error during kernel execution");
	custom_error_check(hipMemcpy(reduce_host, reduce_device, reduce_size, hipMemcpyDeviceToHost), "Failed to copy data FROM device");
	custom_error_check(hipFree(reduce_device), "Failed to free memory on device");
}

__global__ void cuda_do_reduce_fast(DATA_TYPE *grid_device, DATA_TYPE *reduce_device, int nrow, int ncol){
	int idx=blockIdx.x*blockDim.x+threadIdx.x;
	if(idx >= nrow){
		return;
	}
	const int offset = idx * ncol;
	DATA_TYPE val;
	DATA_TYPE sum = 0.0;
	float4 vals;
	for(int i = 0; i < ncol; i += 4){
		vals = *(float4*)(&grid_device[offset+i]);
		val = vals.x + vals.y + vals.z + vals.w;
		sum += val;
	}
	reduce_device[idx] = sum;
}

extern "C" void do_reduce_fast(DATA_TYPE *grid_device, DATA_TYPE *reduce_host, int nrow, int ncol, int block_size){
	long long reduce_size = (long long) nrow * (long long) sizeof(DATA_TYPE);
	DATA_TYPE *reduce_device;
	custom_error_check(hipMalloc((void **) &reduce_device, reduce_size), "Failed to allocate on device");
	dim3 dimBlock(block_size);
	dim3 dimGrid ( ((nrow)/dimBlock.x) + (!((nrow)%dimBlock.x)?0:1) );
	cuda_do_reduce_fast<<<dimGrid,dimBlock>>>(grid_device, reduce_device, nrow, ncol);
	custom_error_check(hipPeekAtLastError(), "Error during kernel execution");
	custom_error_check(hipMemcpy(reduce_host, reduce_device, reduce_size, hipMemcpyDeviceToHost), "Failed to copy data FROM device");
	custom_error_check(hipFree(reduce_device), "Failed to free memory on device");
}

extern "C" void free_on_device(DATA_TYPE *device_ptr){
	custom_error_check(hipFree(device_ptr), "Failed to free memory on device");
}

// Taken from provided sample code
extern "C" void find_best_device() {
	int i,n,best,bestNumberOfMultiprocessors;
	int numberOfCUDAcoresForThisCC=0;
	struct hipDeviceProp_t x;

	if ( hipGetDeviceCount(&n)!=hipSuccess ) {
		//printf("No CUDA-enabled devices were found\n");
	}
	//printf("Found %d CUDA-enabled devices\n",n);
	best=-1;
	bestNumberOfMultiprocessors=-1;
	for (i=0;i<n;i++) {
		hipGetDeviceProperties(&x, i);
		//printf("========================= IDENTITY DATA ==================================\n");
		//printf("GPU model name: %s\n",x.name);
		if (x.integrated==1) {
			//printf("GPU The device is an integrated (motherboard) GPU\n");
		} else {
			//printf("GPU The device is NOT an integrated (motherboard) GPU - i.e. it is a discrete device\n");
		}
		//printf("GPU pciBusID: %d\n",x.pciBusID);
		//printf("GPU pciDeviceID: %d\n",x.pciDeviceID);
		//printf("GPU pciDomainID: %d\n",x.pciDomainID);
		if (x.tccDriver==1) {
			//printf("the device is a Tesla one using TCC driver\n");
		} else {
			//printf("the device is NOT a Tesla one using TCC driver\n");
		}
		//printf("========================= COMPUTE DATA ==================================\n");
		//printf("GPU Compute capability: %d.%d\n",x.major,x.minor);
		switch (x.major) {
			case 1:	// Tesla / T10
				numberOfCUDAcoresForThisCC=8;
				break;
			case 2:	// Fermi
				numberOfCUDAcoresForThisCC=32;
				break;
			case 3:	// Kepler
				numberOfCUDAcoresForThisCC=192;
				break;
			case 5:	// Maxwell
				numberOfCUDAcoresForThisCC=128;
				break;
			case 6:	// Pascal
				switch (x.minor) {
					case 0: // GP100, 64 cuda cores per SM - 7.0 should be prefered over 7.1
						numberOfCUDAcoresForThisCC=64;
						break;
					case 1: // GP102, GP104, GP106, GP107, 128 cuda cores per SM
						numberOfCUDAcoresForThisCC=128;
						break;
					default: // Unknown
						numberOfCUDAcoresForThisCC=0;
						break;
				}
				numberOfCUDAcoresForThisCC=128;
				break;
			case 7:	// Volta
				numberOfCUDAcoresForThisCC=64;
				break;
			default: // Unknown
				numberOfCUDAcoresForThisCC=0;
				break;
		}
		if (x.multiProcessorCount>bestNumberOfMultiprocessors*numberOfCUDAcoresForThisCC) {
			best=i;
			bestNumberOfMultiprocessors=x.multiProcessorCount*numberOfCUDAcoresForThisCC;
		}
		//printf("GPU Clock frequency in hertzs: %d\n",x.clockRate);
		//printf("GPU Device can concurrently copy memory and execute a kernel: %d\n",x.deviceOverlap);
		//printf("GPU number of multi-processors: %d\n",x.multiProcessorCount);
		//printf("GPU maximum number of threads per multi-processor: %d\n",x.maxThreadsPerMultiProcessor);
		//printf("GPU Maximum size of each dimension of a grid: %dx%dx%d\n",x.maxGridSize[0],x.maxGridSize[1],x.maxGridSize[2]);
		//printf("GPU Maximum size of each dimension of a block: %dx%dx%d\n",x.maxThreadsDim[0],x.maxThreadsDim[1],x.maxThreadsDim[2]);
		//printf("GPU Maximum number of threads per block: %d\n",x.maxThreadsPerBlock);
		//printf("GPU Maximum pitch in bytes allowed by memory copies: %u\n",(unsigned int)(x.memPitch));
		//printf("GPU Compute mode is: %d\n",x.computeMode);
		//printf("========================= MEMORY DATA ==================================\n");
		//printf("GPU total global memory: %zu bytes\n",(size_t)(x.totalGlobalMem));
		//printf("GPU peak memory clock frequency in kilohertz: %d bytes\n",x.memoryClockRate);
		//printf("GPU memory bus width: %d bits\n",x.memoryBusWidth);
		//printf("GPU L2 cache size: %d bytes\n",x.l2CacheSize);
		//printf("GPU 32-bit registers available per block: %d\n",x.regsPerBlock);
		//printf("GPU Shared memory available per block in bytes: %d\n",(int)(x.sharedMemPerBlock));
		//printf("GPU Alignment requirement for textures: %d\n",(int)(x.textureAlignment));
		//printf("GPU Constant memory available on device in bytes: %d\n",(int)(x.totalConstMem));
		//printf("GPU Warp size in threads: %d\n",x.warpSize);
		//printf("GPU maximum 1D texture size: %d\n",x.maxTexture1D);
		//printf("GPU maximum 2D texture size: %d %d\n",x.maxTexture2D[0],x.maxTexture2D[1]);
		//printf("GPU maximum 3D texture size: %d %d %d\n",x.maxTexture3D[0],x.maxTexture3D[1],x.maxTexture3D[2]);
		//printf("GPU maximum 1D layered texture dimensions: %d %d\n",x.maxTexture1DLayered[0],x.maxTexture1DLayered[1]);
		//printf("GPU maximum 2D layered texture dimensions: %d %d %d\n",x.maxTexture2DLayered[0],x.maxTexture2DLayered[1],x.maxTexture2DLayered[2]);
		//printf("GPU surface alignment: %d\n",(int)(x.surfaceAlignment));
		if (x.canMapHostMemory==1) {
			//printf("GPU The device can map host memory into the CUDA address space\n");
		} else {
			//printf("GPU The device can NOT map host memory into the CUDA address space\n");
		}
		if (x.ECCEnabled==1) {
			//printf("GPU memory has ECC support\n");
		} else {
			//printf("GPU memory does not have ECC support\n");
		}
		if (x.ECCEnabled==1) {
			//printf("GPU The device shares an unified address space with the host\n");
		} else {

			//printf("GPU The device DOES NOT share an unified address space with the host\n");
		}
		//printf("========================= EXECUTION DATA ==================================\n");
		if (x.concurrentKernels==1) {
			//printf("GPU Concurrent kernels are allowed\n");
		} else {
			//printf("GPU Concurrent kernels are NOT allowed\n");
		}
		if (x.kernelExecTimeoutEnabled==1) {
			//printf("GPU There is a run time limit for kernels executed in the device\n");
		} else {
			//printf("GPU There is NOT a run time limit for kernels executed in the device\n");
		}
		if (x.asyncEngineCount==1) {
			//printf("GPU The device can concurrently copy memory between host and device while executing a kernel\n");
		} else if (x.asyncEngineCount==2) {
			//printf("GPU The device can concurrently copy memory between host and device in both directions and execute a kernel at the same time\n");
		} else {
			//printf("GPU the device is NOT capable of concurrently memory copying\n");
		}
	}
	if (best>=0) {
		hipGetDeviceProperties(&x, best);
		//printf("Choosing %s with %d multiprocessors\n", x.name,bestNumberOfMultiprocessors);
		hipSetDevice(best);
	}
}
